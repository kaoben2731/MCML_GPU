#include "hip/hip_runtime.h"
#include "header.h"
//#include <hip/hip_runtime_api.h>	//YU-modified
//#include <helper_string.h>  //YU-modified
//#include <hip/hip_vector_types.h>	//YU-modified
#include <float.h> //for FLT_MAX
#include <vector> // for vector
#include <assert.h>


int InitMemStructs(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim);
int InitMemStructs_replay(MemStruct_Replay* HostMem, MemStruct_Replay* DeviceMem, SimulationStruct* sim);
void FreeMemStructs(MemStruct* HostMem, MemStruct* DeviceMem);
void FreeMemStructs_replay(MemStruct_Replay* HostMem, MemStruct_Replay* DeviceMem);
void FreeSimulationStruct(SimulationStruct* sim, int n_simulations);
__global__ void MCd(MemStruct DeviceMem, unsigned long long seed);
__global__ void MCd_replay(MemStruct DeviceMem, MemStruct_Replay DeviceMem_Replay, int detected_SDS);
int InitDCMem(SimulationStruct* sim);
int Write_Simulation_Results(MemStruct* HostMem, SimulationStruct* sim, clock_t simulation_time);
int read_simulation_data(char* filename, SimulationStruct** simulations, int ignoreAdetection);
int interpret_arg(int argc, char* argv[], unsigned long long* seed, int* ignoreAdetection);

__global__ void MCd(MemStruct DeviceMem);
__device__ void LaunchPhoton(PhotonStruct* p, hiprandState *state);
__global__ void LaunchPhoton_Global(MemStruct DeviceMem, unsigned long long seed);
__device__ void Spin(PhotonStruct*, float, hiprandState* state);
__device__ unsigned int Reflect(PhotonStruct*, int, hiprandState* state);
__device__ unsigned int PhotonSurvive(PhotonStruct*, hiprandState* state);
__device__ void AtomicAddULL(unsigned long long* address, unsigned int add);
__device__ bool detect(PhotonStruct* p, Fibers* f);
__device__ int binarySearch(float *data, float value);
void fiber_initialization(Fibers* f);
void fiber_initialization_replay(Fibers_Replay* f_r, SimulationStruct* sim);
void output_fiber(SimulationStruct* sim, float* reflectance, char* output); //Wang modified
void output_SDS_pathlength(SimulationStruct* simulation, float ***pathlength_weight_arr, int *temp_SDS_detect_num, int SDS_to_output);
void output_sim_summary(SimulationStruct* simulation, SummaryStruct sumStruc, bool do_replay);
//void calculate_reflectance(Fibers* f, float *result, float (*pathlength_weight_arr)[NUM_LAYER + 1][detected_temp_size], int *total_SDS_detect_num, int *temp_SDS_detect_num);
void calculate_reflectance(Fibers* f, float *result, int* total_SDS_detect_num, vector<vector<hiprandState>>& detected_state_arr);
void calculate_reflectance_replay(Fibers_Replay* f_r, float *result, float ***pathlength_weight_arr, int *temp_SDS_detect_num, int *total_SDS_detect_num, int SDS_should_be, int num_layers);
void input_g(int index, G_Array *g);
int InitG(G_Array* HostG, G_Array* DeviceG, int index);
void FreeG(G_Array* HostG, G_Array* DeviceG);

void output_A_rz(SimulationStruct* sim, unsigned long long *data);
void output_A0_z(SimulationStruct* sim, unsigned long long *data);


__device__ float rn_gen(hiprandState *s)
{
	float x = hiprand_uniform(s);
	return x;
}

void DoOneSimulation(SimulationStruct* simulation, int index, char* output, bool do_replay, bool output_each_pathlength)
{
	SummaryStruct sumStruc;
	sumStruc.time1 = clock(); // tic
	sumStruc.number_of_photons = simulation->number_of_photons;

	vector<vector<hiprandState>> detected_state_arr(simulation->num_detector); // the state for detected photon in hiprand
	int *total_SDS_detect_num = new int[simulation->num_detector]; // record number fo detected photon by each detector
	float *reflectance = new float[simulation->num_detector]; //record reflectance of fibers
	for (int d = 0; d < simulation->num_detector; d++) {
		total_SDS_detect_num[d] = 0;
		reflectance[d] = 0;
	}

	unsigned long long seed = time(NULL);
	srand(seed); // set random seed for main loop
	
	MemStruct DeviceMem;
	MemStruct HostMem;
	unsigned int threads_active_total = 1;
	unsigned int i, ii;

	hipError_t cudastat;

	InitMemStructs(&HostMem, &DeviceMem, simulation);
	InitDCMem(simulation);

	dim3 dimBlock(NUM_THREADS_PER_BLOCK);	printf("NUM_THREADS_PER_BLOCK\t%d\n", NUM_THREADS_PER_BLOCK);
	dim3 dimGrid(NUM_BLOCKS);				printf("NUM_BLOCKS\t%d\n", NUM_BLOCKS);

	LaunchPhoton_Global << <dimGrid, dimBlock >> >(DeviceMem, seed);
	hipDeviceSynchronize(); //CUDA_SAFE_CALL( hipDeviceSynchronize() ); // Wait for all threads to finish
	cudastat = hipGetLastError(); // Check if there was an error
	if (cudastat)printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

	i = 0;

	// run the first time to find the photons can be detected
	while (threads_active_total>0)
	{
		i++;
		fiber_initialization(HostMem.f); //Wang modified
																//printf("Size of Fibers\t%d\n",sizeof(Fibers));
		hipMemcpy(DeviceMem.f, HostMem.f, NUM_THREADS * sizeof(Fibers), hipMemcpyHostToDevice); //malloc sizeof(FIbers) equals to 13*(5*4)

																								  //run the kernel
		seed = rand(); // get seed for MCD
		MCd <<<dimGrid, dimBlock >>>(DeviceMem, seed);
		//cout << "after MCd\n";
		hipDeviceSynchronize(); //CUDA_SAFE_CALL( hipDeviceSynchronize() ); // Wait for all threads to finish
		cudastat = hipGetLastError(); // Check if there was an error
		if (cudastat)printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

		// Copy thread_active from device to host, later deleted
		hipMemcpy(HostMem.thread_active, DeviceMem.thread_active, NUM_THREADS * sizeof(unsigned int), hipMemcpyDeviceToHost); //CUDA_SAFE_CALL(hipMemcpy(HostMem.thread_active,DeviceMem.thread_active,NUM_THREADS*sizeof(unsigned int),hipMemcpyDeviceToHost) );
		threads_active_total = 0;
		for (ii = 0; ii<NUM_THREADS; ii++) threads_active_total += HostMem.thread_active[ii];

		hipMemcpy(HostMem.f, DeviceMem.f, NUM_THREADS * sizeof(Fibers), hipMemcpyDeviceToHost); //CUDA_SAFE_CALL(hipMemcpy(HostMem.f,DeviceMem.f,NUM_THREADS*sizeof(Fibers),hipMemcpyDeviceToHost));
		//cout << "before cal ref\n";
		calculate_reflectance(HostMem.f, reflectance, total_SDS_detect_num, detected_state_arr);
		//cout << "after cal ref\n";

		hipMemcpy(HostMem.num_terminated_photons, DeviceMem.num_terminated_photons, sizeof(unsigned long long), hipMemcpyDeviceToHost);

		printf("\rRun %u, Number of photons terminated %llu, Threads active %u, photon deteced number for SDSs:", i, *HostMem.num_terminated_photons, threads_active_total);
		for (int d = 0; d < simulation->num_detector; d++) {
			printf("\t%d,", total_SDS_detect_num[d]);
		}
		printf("          ");
		//printf("\n");
	}
	sumStruc.time2 = clock(); // toc
	printf("\nfinish first run, cost %.2f secs\n", (double)(sumStruc.time2 - sumStruc.time1) / CLOCKS_PER_SEC);

	sumStruc.total_SDS_detect_num = new int[simulation->num_detector];
	for (int s = 0; s < simulation->num_detector; s++) {
		sumStruc.total_SDS_detect_num[s] = total_SDS_detect_num[s];
	}

	if (!do_replay) { // only output the reflectance
		output_fiber(simulation, reflectance, output);
		output_sim_summary(simulation, sumStruc, do_replay);
	}
	else { // replay the detected photons
		// init the memstruct for replay
		MemStruct_Replay HostMem_Replay;
		MemStruct_Replay DeviceMem_Replay;
		InitMemStructs_replay(&HostMem_Replay, &DeviceMem_Replay, simulation);
		//cout << "after init replay mem\n";

		int *temp_SDS_detect_num = new int[simulation->num_detector]; // record temp number fo detected photon by the detector
		float *replay_reflectance = new float[simulation->num_detector]; //record reflectance of fibers
		//float pathlength_weight_arr[PRESET_NUM_DETECTOR][detected_size][NUM_LAYER + 2]
		float*** pathlength_weight_arr = new float**[simulation->num_detector]; // record the pathlength and weight for each photon, in each layer, and for each detector, also scatter times
		for (int s = 0; s < simulation->num_detector; s++) {
			temp_SDS_detect_num[s] = 0;
			replay_reflectance[s] = 0;
			// init the PL array for this detector
			pathlength_weight_arr[s] = new float*[total_SDS_detect_num[s]]; // record the pathlength and weight for each photon, in each layer, and for each detector, also scatter times
			for (int j = 0; j < total_SDS_detect_num[s]; j++) {
				pathlength_weight_arr[s][j] = new float[simulation->num_layers + 2];
				for (int k = 0; k < simulation->num_layers + 2; k++) {
					pathlength_weight_arr[s][j][k] = 0;
				}
			}

			//printf("\tafter init PL arr for SDS %d\n", s);

			// prepare seeds to copy to device
			int replay_counter = 0;
			while (replay_counter < total_SDS_detect_num[s]) {
				int to_dev_index = 0;
				for (int t = 0; t < NUM_THREADS; t++) {
					HostMem.thread_active[t] = 0;
				}
				while (to_dev_index < NUM_THREADS && replay_counter < total_SDS_detect_num[s]) {
					HostMem.state[to_dev_index] = detected_state_arr[s][replay_counter];
					HostMem.thread_active[to_dev_index] = 1;
					replay_counter++;
					to_dev_index++;
				}
				hipMemcpy(DeviceMem.state, HostMem.state, NUM_THREADS * sizeof(hiprandState), hipMemcpyHostToDevice);
				hipMemcpy(DeviceMem.thread_active, HostMem.thread_active, NUM_THREADS * sizeof(unsigned int), hipMemcpyHostToDevice);
				//printf("\t\tafter copy seeds to device mem\n");

				// init fibers
				fiber_initialization(HostMem.f);
				hipMemcpy(DeviceMem.f, HostMem.f, NUM_THREADS * sizeof(Fibers), hipMemcpyHostToDevice);
				fiber_initialization_replay(HostMem_Replay.f_r, simulation);
				hipMemcpy(DeviceMem_Replay.f_r, HostMem_Replay.f_r, NUM_THREADS * sizeof(Fibers_Replay), hipMemcpyHostToDevice);
				//printf("\t\tafter init fibers\n");

				// replay
				MCd_replay << <dimGrid, dimBlock >> > (DeviceMem, DeviceMem_Replay, s);
				hipDeviceSynchronize(); // Wait for all threads to finish
				cudastat = hipGetLastError(); // Check if there was an error
				if (cudastat)printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));
				//printf("\t\tafter replay\n");

				// process result
				hipMemcpy(HostMem_Replay.f_r, DeviceMem_Replay.f_r, NUM_THREADS * sizeof(Fibers_Replay), hipMemcpyDeviceToHost);
				calculate_reflectance_replay(HostMem_Replay.f_r, replay_reflectance, pathlength_weight_arr, temp_SDS_detect_num, total_SDS_detect_num, s + 1, simulation->num_layers);
				//printf("\t\tafter cal reflectance\n");
			}

		}

		output_SDS_pathlength(simulation, pathlength_weight_arr, temp_SDS_detect_num, 0);
		output_fiber(simulation, replay_reflectance, output);

		// copy the A_rz and A0_z back to host
		hipMemcpy(HostMem_Replay.A_rz, DeviceMem_Replay.A_rz, simulation->num_detector * record_nr * record_nz * sizeof(unsigned long long), hipMemcpyDeviceToHost);
		hipMemcpy(HostMem_Replay.A0_z, DeviceMem_Replay.A0_z, simulation->num_detector * record_nz * sizeof(unsigned long long), hipMemcpyDeviceToHost);

		output_A_rz(simulation, HostMem_Replay.A_rz); // output the absorbance
		output_A0_z(simulation, HostMem_Replay.A0_z);

		sumStruc.time3 = clock();
		printf("finish replay, cost %.2f secs\n", (double)(sumStruc.time3 - sumStruc.time2) / CLOCKS_PER_SEC);
		output_sim_summary(simulation, sumStruc, do_replay);


		// free the memory
		FreeMemStructs_replay(&HostMem_Replay, &DeviceMem_Replay);
		delete[] temp_SDS_detect_num;
		delete[] replay_reflectance;
		for (int i = 0; i < simulation->num_detector; i++) {
			for (int j = 0; j < total_SDS_detect_num[i]; j++) {
				delete[] pathlength_weight_arr[i][j];
			}
			delete[] pathlength_weight_arr[i];
		}
		delete[] pathlength_weight_arr;
	}

	// free the memory
	FreeMemStructs(&HostMem, &DeviceMem);

	delete[] total_SDS_detect_num;
	delete[] reflectance;
}

void calculate_reflectance(Fibers* f, float *result, int* total_SDS_detect_num, vector<vector<hiprandState>>& detected_state_arr)
{
	for (int i = 0; i < NUM_THREADS; i++)
	{
		// record the weight, count detected photon number, and record pathlength
		for (int k = 0; k < f[i].detected_photon_counter; k++) {
			int s = f[i].detected_SDS_number[k]; // the detecting SDS, start from 1
			result[s - 1] += f[i].data[k];
			total_SDS_detect_num[s - 1]++;
			detected_state_arr[s - 1].push_back(f[i].detected_state[k]);
		}
	}
}

//void calculate_reflectance(Fibers* f, float *result, float (*pathlength_weight_arr)[NUM_LAYER + 1][detected_temp_size], int *total_SDS_detect_num, int *temp_SDS_detect_num)
//SDS_should_be: the detecting SDS, start from s=1 for SDS1
void calculate_reflectance_replay(Fibers_Replay* f_r, float *result, float ***pathlength_weight_arr, int *temp_SDS_detect_num, int *total_SDS_detect_num, int SDS_should_be, int num_layers)
{
	for (int i = 0; i < NUM_THREADS; i++)
	{
		// record the weight, count detected photon number, and record pathlength
		result[SDS_should_be -1] += f_r[i].data;
		pathlength_weight_arr[SDS_should_be - 1][temp_SDS_detect_num[SDS_should_be - 1]][0] = f_r[i].data;
		for (int l = 0; l < num_layers; l++) {
			pathlength_weight_arr[SDS_should_be - 1][temp_SDS_detect_num[SDS_should_be - 1]][l + 1] = f_r[i].layer_pathlength[l];
		}
		pathlength_weight_arr[SDS_should_be - 1][temp_SDS_detect_num[SDS_should_be - 1]][num_layers + 1] = f_r[i].scatter_event;
				
		temp_SDS_detect_num[SDS_should_be - 1]++;
		if (temp_SDS_detect_num[SDS_should_be - 1] >= total_SDS_detect_num[SDS_should_be - 1]) { // if all the photons are replayed, then break
			break;
		}
	}
}

//Device function to add an unsigned integer to an unsigned long long using CUDA Compute Capability 1.1
__device__ void AtomicAddULL(unsigned long long* address, unsigned int add)
{
	if (atomicAdd((unsigned int*)address, add) + add<add)
		atomicAdd(((unsigned int*)address) + 1, 1u);
}

__global__ void MCd(MemStruct DeviceMem, unsigned long long seed)
{

	//Block index
	int bx = blockIdx.x;

	//Thread index
	int tx = threadIdx.x;

	//First element processed by the block
	int begin = NUM_THREADS_PER_BLOCK * bx;

	float s;	//step length

	unsigned int w_temp;

	PhotonStruct p = DeviceMem.p[begin + tx];
	Fibers f = DeviceMem.f[begin + tx];

	int new_layer;

	hiprandState state = p.state_run; // get the state of hiprand from the photon

	//First, make sure the thread (photon) is active
	unsigned int ii = 0;
	if (!DeviceMem.thread_active[begin + tx]) ii = NUMSTEPS_GPU;

	bool k = true;

	for (; ii<NUMSTEPS_GPU; ii++) //this is the main while loop
	{
		if (layers_dc[p.layer].mutr != FLT_MAX)
			s = -__logf(rn_gen(&state))*layers_dc[p.layer].mutr;//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
		else
			s = 100.0f;//temporary, say the step in glass is 100 cm.

		//Check for layer transitions and in case, calculate s
		new_layer = p.layer;
		if (p.z + s*p.dz<layers_dc[p.layer].z_min) {
			new_layer--;
			s = __fdividef(layers_dc[p.layer].z_min - p.z, p.dz);
		} //Check for upwards reflection/transmission & calculate new s
		if (p.z + s*p.dz>layers_dc[p.layer].z_max) {
			new_layer++;
			s = __fdividef(layers_dc[p.layer].z_max - p.z, p.dz);
		} //Check for downward reflection/transmission

		p.x += p.dx*s;
		p.y += p.dy*s;
		p.z += p.dz*s;

		p.scatter_event++;

		if (p.z>layers_dc[p.layer].z_max)p.z = layers_dc[p.layer].z_max;//needed?
		if (p.z<layers_dc[p.layer].z_min)p.z = layers_dc[p.layer].z_min;//needed?

		if (new_layer != p.layer)
		{
			int temp_layer = p.layer;
			// set the remaining step length to 0
			s = 0.0f;

			if (Reflect(&p, new_layer, &state) == 0u)//Check for reflection
			{
				if (new_layer == 0)
				{   //Diffuse reflectance
					bool detected = detect(&p, &f);
					if (detected == 1) {
						p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
					}
					else { // maybe the detector will reflect the photon
						if (rn_gen(&state) > *detector_reflectance_dc) {
							p.weight = 0;
						}
						else // reflect into tissue
						{
							p.dz *= -1;
							p.z *= -1;
							p.layer = temp_layer;
						}
					}
				}
				if (new_layer > *n_layers_dc)
				{	//Transmitted
					p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
				}
			}
		}

		if (s > 0.0f)
		{
			// Drop weight (apparently only when the photon is scattered)
			w_temp = __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight));
			//w_temp = layers_dc[p.layer].mua*layers_dc[p.layer].mutr*p.weight;
			p.weight -= w_temp;
			Spin(&p, layers_dc[p.layer].g, &state);
		}

		if (!PhotonSurvive(&p, &state)) //if the photon doesn't survive
		{
			k = false;
			if (atomicAdd(DeviceMem.num_terminated_photons, 1u) < (*num_photons_dc - NUM_THREADS))
			{	// Ok to launch another photon
				LaunchPhoton(&p, &state);//Launch a new photon
				state = p.state_run; // reload the state of this thread
			}
			else
			{	// No more photons should be launched. 
				DeviceMem.thread_active[begin + tx] = 0u; // Set thread to inactive
				ii = NUMSTEPS_GPU;				// Exit main loop
			}
		}

	}//end main for loop!

	p.state_run = state; // store the current hiprand state into the photon

	//// truly need?
	//if (k == true && DeviceMem.thread_active[begin + tx] == 1u)    // photons are not killed after numerous steps
	//{
	//	if (*DeviceMem.num_terminated_photons >= (*num_photons_dc - NUM_THREADS))
	//		DeviceMem.thread_active[begin + tx] = 0u;
	//}

	__syncthreads();//necessary?

	//save the state of the MC simulation in global memory before exiting
	DeviceMem.p[begin + tx] = p;	//This one is incoherent!!!
	DeviceMem.f[begin + tx] = f;

}//end MCd

__global__ void MCd_replay(MemStruct DeviceMem, MemStruct_Replay DeviceMem_Replay, int detected_SDS)
{
	//Block index
	int bx = blockIdx.x;
	//Thread index
	int tx = threadIdx.x;
	//First element processed by the block
	int begin = NUM_THREADS_PER_BLOCK * bx;

	//First, make sure the thread (photon) is active
	if (DeviceMem.thread_active[begin + tx]) {
		float s;	//step length

		unsigned int index, w, index_old;
		index_old = 0;
		w = 0;

		unsigned int w_temp;

		PhotonStruct p = DeviceMem.p[begin + tx];
		Fibers f = DeviceMem.f[begin + tx];
		Fibers_Replay f_r = DeviceMem_Replay.f_r[begin + tx];

		// load the seed of detected photon and launch it
		hiprandState state = DeviceMem.state[begin + tx];
		LaunchPhoton(&p, &state);

		state = p.state_run; // get the state of hiprand from the photon

		int new_layer;

		bool k = true;

		while (true) //this is the main while loop
		{
			if (layers_dc[p.layer].mutr != FLT_MAX)
				s = -__logf(rn_gen(&state))*layers_dc[p.layer].mutr;//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
			else
				s = 100.0f;//temporary, say the step in glass is 100 cm.

			//Check for layer transitions and in case, calculate s
			new_layer = p.layer;
			if (p.z + s*p.dz < layers_dc[p.layer].z_min) {
				new_layer--;
				s = __fdividef(layers_dc[p.layer].z_min - p.z, p.dz);
			} //Check for upwards reflection/transmission & calculate new s
			if (p.z + s*p.dz > layers_dc[p.layer].z_max) {
				new_layer++;
				s = __fdividef(layers_dc[p.layer].z_max - p.z, p.dz);
			} //Check for downward reflection/transmission

			p.x += p.dx*s;
			p.y += p.dy*s;
			p.z += p.dz*s;

			f_r.scatter_event++;
			f_r.layer_pathlength[p.layer - 1] += s;

			if (p.z > layers_dc[p.layer].z_max)p.z = layers_dc[p.layer].z_max;//needed?
			if (p.z < layers_dc[p.layer].z_min)p.z = layers_dc[p.layer].z_min;//needed?

			if (new_layer != p.layer)
			{
				int temp_layer = p.layer;
				// set the remaining step length to 0
				s = 0.0f;

				if (Reflect(&p, new_layer, &state) == 0u)//Check for reflection
				{
					if (new_layer == 0)
					{ //Diffuse reflectance
						bool detected = detect(&p, &f);
						if (detected) { // store the photon information into f_r
							f_r.have_detected = true;
							f_r.data = f.data[0];
							f_r.detected_SDS_number = f.detected_SDS_number[0];

							p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
							break;
						}
						else { // maybe the detector will reflect the photon
							if (rn_gen(&state) > *detector_reflectance_dc) {
								p.weight = 0;
								break;
							}
							else
							{
								p.dz *= -1;
								p.z *= -1;
								p.layer = temp_layer;
							}
						}
					}
					if (new_layer > *n_layers_dc)
					{	//Transmitted
						p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
						break;
					}
				}
			}

			if (s > 0.0f)
			{
				// Drop weight (apparently only when the photon is scattered)
				w_temp = __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight));
				//w_temp = layers_dc[p.layer].mua*layers_dc[p.layer].mutr*p.weight;
				p.weight -= w_temp;

				// store the absorbed power to grid
				if (p.first_scatter) {
					unsigned int index;
					index = detected_SDS*record_nz + min(__float2int_rz(__fdividef(p.z, record_dz)), (int)(record_nz - 1));
					AtomicAddULL(&DeviceMem_Replay.A0_z[index], w_temp);
					p.first_scatter = false;
				}
				else {
					index = detected_SDS*record_nr*record_nz + min(__float2int_rz(__fdividef(p.z, record_dz)), (int)(record_nz - 1)) *record_nr + min(__float2int_rz(__fdividef(sqrtf(p.x*p.x + p.y*p.y), record_dr)), (int)record_nr - 1);

					if (index == index_old)
					{
						w += w_temp;
					}
					else// if(w!=0)
					{
						AtomicAddULL(&DeviceMem_Replay.A_rz[index_old], w);
						index_old = index;
						w = w_temp;
					}
				}

				Spin(&p, layers_dc[p.layer].g, &state);
			}

			if (!PhotonSurvive(&p, &state)) //if the photon doesn't survive
			{
				break;
			}

		}//end main for loop!

		if (w != 0)
			AtomicAddULL(&DeviceMem_Replay.A_rz[index_old], w);

		p.state_run = state; // store the current rand state

		//save the state of the MC simulation in global memory before exiting
		DeviceMem.p[begin + tx] = p;	//This one is incoherent!!!
		DeviceMem.f[begin + tx] = f;
		DeviceMem_Replay.f_r[begin + tx] = f_r;
	}
}//end MCd

__device__ void LaunchPhoton(PhotonStruct* p, hiprandState *state)
{
	p->state_seed = *state; // store the init hiprandState of photon
	/*
	float rnd_position, rnd_Azimuth, rnd_direction, rnd_rotated;
	float AzimuthAngle;
	float launchPosition;
	float theta_direction;
	float rotated_angle;
	float uxprime, uyprime, uzprime;
	float angle = -ANGLE * PI / 180;
	*/


	//rnd_position = rn_gen(state);
	//rnd_Azimuth = rn_gen(state);
	//rnd_direction = rn_gen(state);
	//rnd_rotated = rn_gen(state);
	//AzimuthAngle = 2 * PI * rnd_Azimuth;
	//rotated_angle = 2 * PI * rnd_rotated; //YU-modified // modified to point source

	//float beam_width = 0.0175;  // 175 um, Gaussian beam profile
	//float beam_width = illumination_r;

	// infinite narrow beam for impulse response

	p->x = 0.0;
	p->y = 0.0;
	p->z = 0.0;

	//theta_direction = asin(NAOfSource / n_source)*rnd_direction;
	//p->dz = cos(theta_direction);
	//p->dx = sin(theta_direction) * cos(rotated_angle);
	//p->dy = sin(theta_direction) * sin(rotated_angle);

	//uxprime = cos(angle)*p->dx - sin(angle)*p->dz;
	//uyprime = sin(theta_direction)*sin(rotated_angle);
	//uzprime = sin(angle)*p->dx + cos(angle)*p->dz; // YU-modified

	//p->dx = uxprime, p->dy = uyprime, p->dz = uzprime;
	p->dx = 0.0, p->dy = 0.0, p->dz = 1.0;

	p->layer = 1;
	p->first_scatter = true;

	p->scatter_event = 0;
	
	p->weight = *start_weight_dc; //specular reflection!
	p->state_run = *state; // store the current state of photon after serveral rand
}


__global__ void LaunchPhoton_Global(MemStruct DeviceMem, unsigned long long seed)
{
	int bx = blockIdx.x;
	int tx = threadIdx.x;

	//First element processed by the block
	int begin = NUM_THREADS_PER_BLOCK*bx;

	PhotonStruct p;

	hiprandState state = DeviceMem.state[begin + tx];
	hiprand_init(seed, begin+tx, 0, &state); // init hiprandState for each photon

	LaunchPhoton(&p, &state);

	//__syncthreads();//necessary?
	DeviceMem.p[begin + tx] = p;//incoherent!?
}


__device__ void Spin(PhotonStruct* p, float g, hiprandState *state)
{
	float theta, cost, sint;	// cosine and sine of the 
								// polar deflection angle theta. 
	float cosp, sinp;	// cosine and sine of the 
						// azimuthal angle psi. 
	float temp;
	float tempdir = p->dx;

	//This is more efficient for g!=0 but of course less efficient for g==0
	temp = __fdividef((1.0f - (g)*(g)), (1.0f - (g)+2.0f*(g)*rn_gen(state)));//Should be close close????!!!!!
	cost = __fdividef((1.0f + (g)*(g)-temp*temp), (2.0f*(g)));
	if (g == 0.0f)
		cost = 2.0f*rn_gen(state) - 1.0f;//Should be close close??!!!!!

	sint = sqrtf(1.0f - cost*cost);

	__sincosf(2.0f*PI*rn_gen(state), &sinp, &cosp);// spin psi [0-2*PI)

	temp = sqrtf(1.0f - p->dz*p->dz);

	if (temp == 0.0f) //normal incident.
	{
		p->dx = sint*cosp;
		p->dy = sint*sinp;
		p->dz = copysignf(cost, p->dz*cost);
	}
	else // regular incident.
	{
		p->dx = __fdividef(sint*(p->dx*p->dz*cosp - p->dy*sinp), temp) + p->dx*cost;
		p->dy = __fdividef(sint*(p->dy*p->dz*cosp + tempdir*sinp), temp) + p->dy*cost;
		p->dz = -sint*cosp*temp + p->dz*cost;
	}

	//normalisation seems to be required as we are using floats! Otherwise the small numerical error will accumulate
	temp = rsqrtf(p->dx*p->dx + p->dy*p->dy + p->dz*p->dz);
	p->dx = p->dx*temp;
	p->dy = p->dy*temp;
	p->dz = p->dz*temp;
}// end Spin



__device__ unsigned int Reflect(PhotonStruct* p, int new_layer, hiprandState *state)
{
	//Calculates whether the photon is reflected (returns 1) or not (returns 0)
	// Reflect() will also update the current photon layer (after transmission) and photon direction (both transmission and reflection)

	float n1 = layers_dc[p->layer].n;
	float n2 = layers_dc[new_layer].n;
	float r;
	float cos_angle_i = fabsf(p->dz);

	if (n1 == n2)//refraction index matching automatic transmission and no direction change
	{
		p->layer = new_layer;
		return 0u;
	}

	if (n1>n2 && n2*n2<n1*n1*(1 - cos_angle_i*cos_angle_i))//total internal reflection, no layer change but z-direction mirroring
	{
		p->dz *= -1.0f;
		return 1u;
	}

	if (cos_angle_i == 1.0f)//normal incident
	{
		r = __fdividef((n1 - n2), (n1 + n2));
		if (rn_gen(state) <= r*r)
		{
			//reflection, no layer change but z-direction mirroring
			p->dz *= -1.0f;
			return 1u;
		}
		else
		{	//transmission, no direction change but layer change
			p->layer = new_layer;
			return 0u;
		}
	}

	//gives almost exactly the same results as the old MCML way of doing the calculation but does it slightly faster
	// save a few multiplications, calculate cos_angle_i^2;
	float e = __fdividef(n1*n1, n2*n2)*(1.0f - cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
	r = 2 * sqrtf((1.0f - cos_angle_i*cos_angle_i)*(1.0f - e)*e*cos_angle_i*cos_angle_i);//use r as a temporary variable
	e = e + (cos_angle_i*cos_angle_i)*(1.0f - 2.0f*e);//Update the value of e
	r = e*__fdividef((1.0f - e - r), ((1.0f - e + r)*(e + r)));//Calculate r	

	if (rn_gen(state) <= r)
	{
		// Reflection, mirror z-direction!
		p->dz *= -1.0f;
		return 1u;
	}
	else
	{
		// Transmission, update layer and direction
		r = __fdividef(n1, n2);
		e = r*r*(1.0f - cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
		p->dx *= r;
		p->dy *= r;
		p->dz = copysignf(sqrtf(1 - e), p->dz);
		p->layer = new_layer;
		return 0u;
	}

}

__device__ unsigned int PhotonSurvive(PhotonStruct* p, hiprandState *state)
{
	//Calculate wether the photon survives (returns 1) or dies (returns 0)
	if (p->scatter_event >= max_scatter_time) return 0; // scatter too many times, terminate the photon

	if (p->weight>WEIGHTI) return 1u; // No roulette needed
	if (p->weight == 0u) return 0u;	// Photon has exited slab, i.e. kill the photon

	if (rn_gen(state) < CHANCE)
	{
		p->weight = __float2uint_rn(__fdividef((float)p->weight, CHANCE));
		//p->weight = __fdividef((float)p->weight,CHANCE);
		return 1u;
	}
	return 0u;
}

__device__ bool detect(PhotonStruct* p, Fibers* f)
{
	float angle = ANGLE*PI / 180; //YU-modified
	float uz_rotated = (p->dx*sin(angle)) + (p->dz*cos(angle)); //YU-modified
	float uz_angle = acos(fabs(uz_rotated)); //YU-modified
	float distance;
	bool detected_flag=false;

	distance = sqrt(p->x * p->x + p->y * p->y);
	
	for (int i = 1; i <= *num_detector_dc; i++)
	{
		if (uz_angle <= critical_angle_dc[i]) { // successfully detected
			if ((distance >= (detInfo_dc[i].position - detInfo_dc[i].raduis)) && (distance <= (detInfo_dc[i].position + detInfo_dc[i].raduis))) {
				float temp;
				temp = (distance*distance + detInfo_dc[i].position * detInfo_dc[i].position - detInfo_dc[i].raduis * detInfo_dc[i].raduis) / (2 * distance * detInfo_dc[i].position);
				// check for rounding error!
				if (temp > 1.0f)
					temp = 1.0f;

				if (f->detected_photon_counter < SDS_detected_temp_size) {
					f->detected_SDS_number[f->detected_photon_counter] = i;
					f->data[f->detected_photon_counter] = p->weight  * acos(temp) * RPI;
					f->detected_state[f->detected_photon_counter] = p->state_seed;
					f->detected_photon_counter++;
				}
				detected_flag = true;
			}
		}
	}
	if (detected_flag) {
		return true;
	}
	else {
		return false;
	}
}

int InitDCMem(SimulationStruct* sim)
{
	// Copy n_layers_dc to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(n_layers_dc), &(sim->num_layers), sizeof(unsigned int));

	// Copy num_detector_dc to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(num_detector_dc), &(sim->num_detector), sizeof(unsigned int));

	// Copy start_weight_dc to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(start_weight_dc), &(sim->start_weight), sizeof(float));

	// Copy layer data to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(layers_dc), sim->layers, (sim->num_layers + 2) * sizeof(LayerStruct));

	// Copy detector data to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(detInfo_dc), sim->detInfo, (sim->num_detector + 1) * sizeof(DetectorInfoStruct));

	// Copy num_photons_dc to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(num_photons_dc), &(sim->number_of_photons), sizeof(unsigned long long));
	
	// Copy detector_reflectance_dc to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(detector_reflectance_dc), &(sim->detector_reflectance), sizeof(float));

	// Copy detector critical angles to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(critical_angle_dc), sim->critical_arr, (sim->num_detector + 1) * sizeof(float));
	

	return 0;
}

int InitMemStructs(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim) //Wang modified
{
	// Allocate p on the device!!
	hipMalloc((void**)&DeviceMem->p, NUM_THREADS * sizeof(PhotonStruct));

	// Allocate thread_active on the device and host
	HostMem->thread_active = (unsigned int*)malloc(NUM_THREADS * sizeof(unsigned int));
	if (HostMem->thread_active == NULL) { printf("Error allocating HostMem->thread_active"); exit(1); }
	for (int i = 0; i<NUM_THREADS; i++)HostMem->thread_active[i] = 1u;

	hipMalloc((void**)&DeviceMem->thread_active, NUM_THREADS * sizeof(unsigned int));
	hipMemcpy(DeviceMem->thread_active, HostMem->thread_active, NUM_THREADS * sizeof(unsigned int), hipMemcpyHostToDevice);

	//Allocate num_launched_photons on the device and host
	HostMem->num_terminated_photons = (unsigned long long*) malloc(sizeof(unsigned long long));
	if (HostMem->num_terminated_photons == NULL) { printf("Error allocating HostMem->num_terminated_photons"); exit(1); }
	*HostMem->num_terminated_photons = 0;

	hipMalloc((void**)&DeviceMem->num_terminated_photons, sizeof(unsigned long long));
	hipMemcpy(DeviceMem->num_terminated_photons, HostMem->num_terminated_photons, sizeof(unsigned long long), hipMemcpyHostToDevice);

	//Allocate and initialize fiber f on the device and host
	HostMem->f = (Fibers*)malloc(NUM_THREADS * sizeof(Fibers));
	hipMalloc((void**)&DeviceMem->f, NUM_THREADS * sizeof(Fibers));
	fiber_initialization(HostMem->f); //Wang modified
	hipMemcpy(DeviceMem->f, HostMem->f, NUM_THREADS * sizeof(Fibers), hipMemcpyHostToDevice);

	//Allocate states on the device and host
	HostMem->state = (hiprandState*)malloc(NUM_THREADS * sizeof(hiprandState));
	hipMalloc((void**)&DeviceMem->state, NUM_THREADS * sizeof(hiprandState));

	return 1;
}

int InitMemStructs_replay(MemStruct_Replay* HostMem, MemStruct_Replay* DeviceMem, SimulationStruct* sim)
{
	//Allocate and initialize fiber_Replay f_r on the device and host
	HostMem->f_r = (Fibers_Replay*)malloc(NUM_THREADS * sizeof(Fibers_Replay));
	hipMalloc((void**)&DeviceMem->f_r, NUM_THREADS * sizeof(Fibers_Replay));
	fiber_initialization_replay(HostMem->f_r, sim);
	hipMemcpy(DeviceMem->f_r, HostMem->f_r, NUM_THREADS * sizeof(Fibers_Replay), hipMemcpyHostToDevice);

	int rz_size, z0_size;
	rz_size = sim->num_detector*record_nr*record_nz;
	z0_size = sim->num_detector*record_nz;

	// Allocate A_rz on host and device
	HostMem->A_rz = (unsigned long long*) malloc(rz_size * sizeof(unsigned long long));
	if (HostMem->A_rz == NULL) { printf("Error allocating HostMem->A_rz"); exit(1); }
	hipMalloc((void**)&DeviceMem->A_rz, rz_size * sizeof(unsigned long long));
	hipMemset(DeviceMem->A_rz, 0, rz_size * sizeof(unsigned long long));

	// Allocate A0_z on host and device
	HostMem->A0_z = (unsigned long long*) malloc(z0_size * sizeof(unsigned long long));
	if (HostMem->A0_z == NULL) { printf("Error allocating HostMem->A_rz"); exit(1); }
	hipMalloc((void**)&DeviceMem->A0_z, z0_size * sizeof(unsigned long long));
	hipMemset(DeviceMem->A0_z, 0, z0_size * sizeof(unsigned long long));

	return 1;
}

void FreeMemStructs(MemStruct* HostMem, MemStruct* DeviceMem)
{
	free(HostMem->thread_active);
	free(HostMem->num_terminated_photons);
	free(HostMem->f);
	free(HostMem->state);

	hipFree(DeviceMem->thread_active);
	hipFree(DeviceMem->num_terminated_photons);
	hipFree(DeviceMem->f);
	hipFree(DeviceMem->state);
}

void FreeMemStructs_replay(MemStruct_Replay* HostMem, MemStruct_Replay* DeviceMem)
{
	free(HostMem->f_r);
	free(HostMem->A_rz);
	free(HostMem->A0_z);

	hipFree(DeviceMem->f_r);
	hipFree(DeviceMem->A_rz);
	hipFree(DeviceMem->A0_z);
}