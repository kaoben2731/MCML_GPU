#include "hip/hip_runtime.h"
#include "header.h"
//#include <hip/hip_runtime_api.h>	//YU-modified
//#include <helper_string.h>  //YU-modified
//#include <hip/hip_vector_types.h>	//YU-modified
#include <float.h> //for FLT_MAX


int InitMemStructs(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim, char* fiber1_position); //Wang modified
void FreeMemStructs(MemStruct* HostMem, MemStruct* DeviceMem);
void FreeSimulationStruct(SimulationStruct* sim, int n_simulations);
__global__ void MCd(MemStruct DeviceMem, unsigned long long seed);
__global__ void LaunchPhoton_Global(MemStruct DeviceMem);
int InitDCMem(SimulationStruct* sim);
int Write_Simulation_Results(MemStruct* HostMem, SimulationStruct* sim, clock_t simulation_time);
int read_simulation_data(char* filename, SimulationStruct** simulations, int ignoreAdetection);
int interpret_arg(int argc, char* argv[], unsigned long long* seed, int* ignoreAdetection);

__global__ void MCd(MemStruct DeviceMem);
__device__ void LaunchPhoton(PhotonStruct* p, hiprandState *state);
__global__ void LaunchPhoton_Global(MemStruct DeviceMem, unsigned long long seed);
__device__ void Spin(PhotonStruct*, float, hiprandState* state);
__device__ unsigned int Reflect(PhotonStruct*, int, hiprandState* state);
__device__ unsigned int PhotonSurvive(PhotonStruct*, hiprandState* state);
__device__ void AtomicAddULL(unsigned long long* address, unsigned int add);
__device__ void detect(PhotonStruct* p, Fibers* f);
__device__ int binarySearch(float *data, float value);
void fiber_initialization(Fibers* f, float fiber1_position); //Wang modified
void output_fiber(SimulationStruct* sim, float* reflectance, char* output); //Wang modified
void output_SDS_pathlength(float ***pathlength_weight_arr, int *temp_SDS_detect_num, int SDS_to_output);
void output_sim_summary(SimulationStruct* sim, int *total_SDS_detect_num);
//void calculate_reflectance(Fibers* f, float *result, float (*pathlength_weight_arr)[NUM_LAYER + 1][detected_num_total], int *total_SDS_detect_num, int *temp_SDS_detect_num);
void calculate_reflectance(Fibers* f, float *result, float ***pathlength_weight_arr, int *total_SDS_detect_num, int *temp_SDS_detect_num);
void input_g(int index, G_Array *g);
int InitG(G_Array* HostG, G_Array* DeviceG, int index);
void FreeG(G_Array* HostG, G_Array* DeviceG);


__device__ float rn_gen(hiprandState *s)
{
	float x = hiprand_uniform(s);
	return x;
}

void DoOneSimulation(SimulationStruct* simulation, int index, char* output, char* fiber1_position) //Wang modified
{
	printf("to here\n");
	unsigned long long seed = time(NULL);
	float reflectance[NUM_OF_DETECTOR] = { 0 }; //record reflectance of fibers
	//float pathlength_weight_arr[NUM_OF_DETECTOR][NUM_LAYER + 1][detected_num_total]
	float*** pathlength_weight_arr = new float**[NUM_OF_DETECTOR]; // record the pathlength and weight for each photon, in each layer, and for each detector
	for (int i = 0; i < NUM_OF_DETECTOR; i++) {
		pathlength_weight_arr[i] = new float*[NUM_LAYER + 1];
		for (int j = 0; j < NUM_LAYER + 1; j++) {
			pathlength_weight_arr[i][j] = new float[detected_num_total];
			for (int k = 0; k < detected_num_total; k++) {
				pathlength_weight_arr[i][j][k] = 0;
			}
		}
	}

	int total_SDS_detect_num[NUM_OF_DETECTOR] = { 0 }; // record number fo detected photon by each detector
	int temp_SDS_detect_num[NUM_OF_DETECTOR] = { 0 }; // record temp number fo detected photon by each detector, for prevent the collected photon number exceed detected_num_total

	cout << "to here\n";

	MemStruct DeviceMem;
	MemStruct HostMem;
	unsigned int threads_active_total = 1;
	unsigned int i, ii;

	hipError_t cudastat;

	InitMemStructs(&HostMem, &DeviceMem, simulation, fiber1_position); //Wang modified
	InitDCMem(simulation);

	dim3 dimBlock(NUM_THREADS_PER_BLOCK);	printf("NUM_THREADS_PER_BLOCK\t%d\n", NUM_THREADS_PER_BLOCK);
	dim3 dimGrid(NUM_BLOCKS);				printf("NUM_BLOCKS\t%d\n", NUM_BLOCKS);

	LaunchPhoton_Global << <dimGrid, dimBlock >> >(DeviceMem, seed);
	hipDeviceSynchronize(); //CUDA_SAFE_CALL( hipDeviceSynchronize() ); // Wait for all threads to finish
	cudastat = hipGetLastError(); // Check if there was an error
	if (cudastat)printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

	i = 0;

	cout << "to here\n";

	while (threads_active_total>0)
	{
		i++;
		fiber_initialization(HostMem.f, atof(fiber1_position)); //Wang modified
																//printf("Size of Fibers\t%d\n",sizeof(Fibers));
		hipMemcpy(DeviceMem.f, HostMem.f, NUM_THREADS * sizeof(Fibers), hipMemcpyHostToDevice); //malloc sizeof(FIbers) equals to 13*(5*4)

																								  //run the kernel
		seed = time(NULL);
		MCd << <dimGrid, dimBlock >> >(DeviceMem, seed);
		hipDeviceSynchronize(); //CUDA_SAFE_CALL( hipDeviceSynchronize() ); // Wait for all threads to finish
		cudastat = hipGetLastError(); // Check if there was an error
		if (cudastat)printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

		// Copy thread_active from device to host, later deleted
		hipMemcpy(HostMem.thread_active, DeviceMem.thread_active, NUM_THREADS * sizeof(unsigned int), hipMemcpyDeviceToHost); //CUDA_SAFE_CALL(hipMemcpy(HostMem.thread_active,DeviceMem.thread_active,NUM_THREADS*sizeof(unsigned int),hipMemcpyDeviceToHost) );
		threads_active_total = 0;
		for (ii = 0; ii<NUM_THREADS; ii++) threads_active_total += HostMem.thread_active[ii];

		hipMemcpy(HostMem.f, DeviceMem.f, NUM_THREADS * sizeof(Fibers), hipMemcpyDeviceToHost); //CUDA_SAFE_CALL(hipMemcpy(HostMem.f,DeviceMem.f,NUM_THREADS*sizeof(Fibers),hipMemcpyDeviceToHost));
		calculate_reflectance(HostMem.f, reflectance, pathlength_weight_arr, total_SDS_detect_num, temp_SDS_detect_num);

		hipMemcpy(HostMem.num_terminated_photons, DeviceMem.num_terminated_photons, sizeof(unsigned int), hipMemcpyDeviceToHost);

		printf("Run %u, Number of photons terminated %u, Threads active %u, photon deteced number for SDSs:", i, *HostMem.num_terminated_photons, threads_active_total);
		for (int d = 0; d < NUM_OF_DETECTOR; d++) {
			printf("\t%d,", total_SDS_detect_num[d]);
		}
		printf("\n");
	}
	//cout << "#" << index << " Simulation done!\n";

	output_SDS_pathlength(pathlength_weight_arr, temp_SDS_detect_num, 0);
	output_fiber(simulation, reflectance, output);
	output_sim_summary(simulation, total_SDS_detect_num);

	// free the memory
	FreeMemStructs(&HostMem, &DeviceMem);

	for (int i = 0; i < NUM_OF_DETECTOR; i++) {
		for (int j = 0; j < NUM_LAYER + 1; j++) {
			delete[] pathlength_weight_arr[i][j];
		}
		delete[] pathlength_weight_arr[i];
	}
	delete[] pathlength_weight_arr;
}

//void calculate_reflectance(Fibers* f, float *result, float (*pathlength_weight_arr)[NUM_LAYER + 1][detected_num_total], int *total_SDS_detect_num, int *temp_SDS_detect_num)
void calculate_reflectance(Fibers* f, float *result, float ***pathlength_weight_arr, int *total_SDS_detect_num, int *temp_SDS_detect_num)
{
	for (int i = 0; i < NUM_THREADS; i++)
	{
		if (NORMAL)
		{
			for (int k = 1; k <= NUM_OF_DETECTOR; k++) {
				result[k - 1] += f[i].data[k];
				/*result[0] += f[i].data[1];
				result[1] += f[i].data[2];
				result[2] += f[i].data[3];
				result[3] += f[i].data[4];*/ //YU-modified
			}
		}
		else
		{   //Wang-modified

			for (int k = 1; k <= NUM_OF_DETECTOR; k++) {
				if (f[i].photon_detected[k]) {
					// record the weight, count detected photon number, and record pathlength
					result[k - 1] += f[i].data[k];
					
					pathlength_weight_arr[k - 1][0][temp_SDS_detect_num[k - 1]] = f[i].data[k];
					for (int l = 0; l < NUM_LAYER; l++) {
						pathlength_weight_arr[k - 1][l + 1][temp_SDS_detect_num[k - 1]] = f[i].layer_pathlength[l];
					}
					
					temp_SDS_detect_num[k - 1]++;
					total_SDS_detect_num[k - 1]++;

					if (temp_SDS_detect_num[k - 1] >= detected_num_total) {
						output_SDS_pathlength(pathlength_weight_arr, temp_SDS_detect_num, k);
					}
				}
			}
		}
	}
}

//Device function to add an unsigned integer to an unsigned long long using CUDA Compute Capability 1.1
__device__ void AtomicAddULL(unsigned long long* address, unsigned int add)
{
	if (atomicAdd((unsigned int*)address, add) + add<add)
		atomicAdd(((unsigned int*)address) + 1, 1u);
}

__global__ void MCd(MemStruct DeviceMem, unsigned long long seed)
{
	//Block index
	int bx = blockIdx.x;

	//Thread index
	int tx = threadIdx.x;

	//First element processed by the block
	int begin = NUM_THREADS_PER_BLOCK * bx;

	float s;	//step length

	unsigned int w_temp;

	PhotonStruct p = DeviceMem.p[begin + tx];
	Fibers f = DeviceMem.f[begin + tx];

	int new_layer;

	hiprandState state = DeviceMem.state[begin + tx];
	hiprand_init(seed, begin + tx, 0, &state);

	//First, make sure the thread (photon) is active
	unsigned int ii = 0;
	if (!DeviceMem.thread_active[begin + tx]) ii = NUMSTEPS_GPU;

	bool k = true;

	for (; ii<NUMSTEPS_GPU; ii++) //this is the main while loop
	{
		if (layers_dc[p.layer].mutr != FLT_MAX)
			s = -__logf(rn_gen(&state))*layers_dc[p.layer].mutr;//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
		else
			s = 100.0f;//temporary, say the step in glass is 100 cm.

		//Check for layer transitions and in case, calculate s
		new_layer = p.layer;
		if (p.z + s*p.dz<layers_dc[p.layer].z_min) {
			new_layer--;
			s = __fdividef(layers_dc[p.layer].z_min - p.z, p.dz);
		} //Check for upwards reflection/transmission & calculate new s
		if (p.z + s*p.dz>layers_dc[p.layer].z_max) {
			new_layer++;
			s = __fdividef(layers_dc[p.layer].z_max - p.z, p.dz);
		} //Check for downward reflection/transmission

		p.x += p.dx*s;
		p.y += p.dy*s;
		p.z += p.dz*s;

		f.layer_pathlength[p.layer-1] += s;

		if (p.z>layers_dc[p.layer].z_max)p.z = layers_dc[p.layer].z_max;//needed?
		if (p.z<layers_dc[p.layer].z_min)p.z = layers_dc[p.layer].z_min;//needed?

		if (new_layer != p.layer)
		{
			// set the remaining step length to 0
			s = 0.0f;

			if (Reflect(&p, new_layer, &state) == 0u)//Check for reflection
			{
				if (new_layer == 0)
				{ //Diffuse reflectance
					detect(&p, &f);
					p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
				}
				if (new_layer > *n_layers_dc)
				{	//Transmitted
					p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
				}
			}
		}

		if (s > 0.0f)
		{
			// Drop weight (apparently only when the photon is scattered)
			w_temp = __float2uint_rn(layers_dc[p.layer].mua*layers_dc[p.layer].mutr*__uint2float_rn(p.weight));
			//w_temp = layers_dc[p.layer].mua*layers_dc[p.layer].mutr*p.weight;
			p.weight -= w_temp;

			Spin(&p, layers_dc[p.layer].g, &state);
		}

		if (!PhotonSurvive(&p, &state)) //if the photon doesn't survive
		{
			k = false;
			if (atomicAdd(DeviceMem.num_terminated_photons, 1u) < (*num_photons_dc - NUM_THREADS))
			{	// Ok to launch another photon
				LaunchPhoton(&p, &state);//Launch a new photon
			}
			else
			{	// No more photons should be launched. 
				DeviceMem.thread_active[begin + tx] = 0u; // Set thread to inactive
				ii = NUMSTEPS_GPU;				// Exit main loop
			}
		}

	}//end main for loop!

	if (k == true && DeviceMem.thread_active[begin + tx] == 1u)    // photons are not killed after numerous steps
	{
		if (atomicAdd(DeviceMem.num_terminated_photons, 1u) >= (*num_photons_dc - NUM_THREADS))
			DeviceMem.thread_active[begin + tx] = 0u;
	}

	__syncthreads();//necessary?

					//save the state of the MC simulation in global memory before exiting
	DeviceMem.p[begin + tx] = p;	//This one is incoherent!!!
	DeviceMem.f[begin + tx] = f;

}//end MCd

__device__ void LaunchPhoton(PhotonStruct* p, hiprandState *state)
{
	float rnd_position, rnd_Azimuth, rnd_direction, rnd_rotated;
	float AzimuthAngle;
	float launchPosition;
	float theta_direction;
	float rotated_angle;
	float uxprime, uyprime, uzprime;
	float angle = -ANGLE * PI / 180;


	//rnd_position = rn_gen(state);
	//rnd_Azimuth = rn_gen(state);
	//rnd_direction = rn_gen(state);
	//rnd_rotated = rn_gen(state);
	//AzimuthAngle = 2 * PI * rnd_Azimuth;
	//rotated_angle = 2 * PI * rnd_rotated; //YU-modified // modified to point source

	//float beam_width = 0.0175;  // 175 um, Gaussian beam profile
	//float beam_width = illumination_r;

	// infinite narrow beam for impulse response

	p->x = 0.0;
	p->y = 0.0;
	p->z = 0.0;

	//theta_direction = asin(NAOfSource / n_source)*rnd_direction;
	//p->dz = cos(theta_direction);
	//p->dx = sin(theta_direction) * cos(rotated_angle);
	//p->dy = sin(theta_direction) * sin(rotated_angle);

	//uxprime = cos(angle)*p->dx - sin(angle)*p->dz;
	//uyprime = sin(theta_direction)*sin(rotated_angle);
	//uzprime = sin(angle)*p->dx + cos(angle)*p->dz; // YU-modified

	//p->dx = uxprime, p->dy = uyprime, p->dz = uzprime;
	p->dx = 0.0, p->dy = 0.0, p->dz = 1.0;

	p->layer = 1;
	p->first_scatter = true;

	p->weight = *start_weight_dc; //specular reflection!
}


__global__ void LaunchPhoton_Global(MemStruct DeviceMem, unsigned long long seed)
{
	int bx = blockIdx.x;
	int tx = threadIdx.x;

	//First element processed by the block
	int begin = NUM_THREADS_PER_BLOCK*bx;

	PhotonStruct p;

	hiprandState state = DeviceMem.state[begin + tx];
	hiprand_init(seed, 0, 0, &state);

	LaunchPhoton(&p, &state);

	//__syncthreads();//necessary?
	DeviceMem.p[begin + tx] = p;//incoherent!?
}


__device__ void Spin(PhotonStruct* p, float g, hiprandState *state)
{
	float theta, cost, sint;	// cosine and sine of the 
								// polar deflection angle theta. 
	float cosp, sinp;	// cosine and sine of the 
						// azimuthal angle psi. 
	float temp;
	float tempdir = p->dx;

	//This is more efficient for g!=0 but of course less efficient for g==0
	temp = __fdividef((1.0f - (g)*(g)), (1.0f - (g)+2.0f*(g)*rn_gen(state)));//Should be close close????!!!!!
	cost = __fdividef((1.0f + (g)*(g)-temp*temp), (2.0f*(g)));
	if (g == 0.0f)
		cost = 2.0f*rn_gen(state) - 1.0f;//Should be close close??!!!!!

	sint = sqrtf(1.0f - cost*cost);

	__sincosf(2.0f*PI*rn_gen(state), &sinp, &cosp);// spin psi [0-2*PI)

	temp = sqrtf(1.0f - p->dz*p->dz);

	if (temp == 0.0f) //normal incident.
	{
		p->dx = sint*cosp;
		p->dy = sint*sinp;
		p->dz = copysignf(cost, p->dz*cost);
	}
	else // regular incident.
	{
		p->dx = __fdividef(sint*(p->dx*p->dz*cosp - p->dy*sinp), temp) + p->dx*cost;
		p->dy = __fdividef(sint*(p->dy*p->dz*cosp + tempdir*sinp), temp) + p->dy*cost;
		p->dz = -sint*cosp*temp + p->dz*cost;
	}

	//normalisation seems to be required as we are using floats! Otherwise the small numerical error will accumulate
	temp = rsqrtf(p->dx*p->dx + p->dy*p->dy + p->dz*p->dz);
	p->dx = p->dx*temp;
	p->dy = p->dy*temp;
	p->dz = p->dz*temp;
}// end Spin



__device__ unsigned int Reflect(PhotonStruct* p, int new_layer, hiprandState *state)
{
	//Calculates whether the photon is reflected (returns 1) or not (returns 0)
	// Reflect() will also update the current photon layer (after transmission) and photon direction (both transmission and reflection)

	float n1 = layers_dc[p->layer].n;
	float n2 = layers_dc[new_layer].n;
	float r;
	float cos_angle_i = fabsf(p->dz);

	if (n1 == n2)//refraction index matching automatic transmission and no direction change
	{
		p->layer = new_layer;
		return 0u;
	}

	if (n1>n2 && n2*n2<n1*n1*(1 - cos_angle_i*cos_angle_i))//total internal reflection, no layer change but z-direction mirroring
	{
		p->dz *= -1.0f;
		return 1u;
	}

	if (cos_angle_i == 1.0f)//normal incident
	{
		r = __fdividef((n1 - n2), (n1 + n2));
		if (rn_gen(state) <= r*r)
		{
			//reflection, no layer change but z-direction mirroring
			p->dz *= -1.0f;
			return 1u;
		}
		else
		{	//transmission, no direction change but layer change
			p->layer = new_layer;
			return 0u;
		}
	}

	//gives almost exactly the same results as the old MCML way of doing the calculation but does it slightly faster
	// save a few multiplications, calculate cos_angle_i^2;
	float e = __fdividef(n1*n1, n2*n2)*(1.0f - cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
	r = 2 * sqrtf((1.0f - cos_angle_i*cos_angle_i)*(1.0f - e)*e*cos_angle_i*cos_angle_i);//use r as a temporary variable
	e = e + (cos_angle_i*cos_angle_i)*(1.0f - 2.0f*e);//Update the value of e
	r = e*__fdividef((1.0f - e - r), ((1.0f - e + r)*(e + r)));//Calculate r	

	if (rn_gen(state) <= r)
	{
		// Reflection, mirror z-direction!
		p->dz *= -1.0f;
		return 1u;
	}
	else
	{
		// Transmission, update layer and direction
		r = __fdividef(n1, n2);
		e = r*r*(1.0f - cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
		p->dx *= r;
		p->dy *= r;
		p->dz = copysignf(sqrtf(1 - e), p->dz);
		p->layer = new_layer;
		return 0u;
	}

}

__device__ unsigned int PhotonSurvive(PhotonStruct* p, hiprandState *state)
{
	//Calculate wether the photon survives (returns 1) or dies (returns 0)

	if (p->weight>WEIGHTI) return 1u; // No roulette needed
	if (p->weight == 0u) return 0u;	// Photon has exited slab, i.e. kill the photon

	if (rn_gen(state) < CHANCE)
	{
		p->weight = __float2uint_rn(__fdividef((float)p->weight, CHANCE));
		//p->weight = __fdividef((float)p->weight,CHANCE);
		return 1u;
	}
	return 0u;
}

__device__ void detect(PhotonStruct* p, Fibers* f)
{
	float angle = ANGLE*PI / 180; //YU-modified
	float critical = asin(f->NA[1] / n_detector); //YU-modified
	float uz_rotated = (p->dx*sin(angle)) + (p->dz*cos(angle)); //YU-modified
	float uz_angle = acos(fabs(uz_rotated)); //YU-modified
	float distance;

	// NA consideration
	if (uz_angle <= critical)  // successfully detected
	{
		if (NORMAL)
		{
			distance = sqrt(p->x * p->x + p->y * p->y);
			if (distance >= 0.025 && distance <= 0.035)           // SDS = 0.03 cm
				f->data[1] += p->weight / 6.0;
			if (distance >= 0.03 && distance <= 0.05)             // SDS = 0.04 cm
				f->data[2] += p->weight / 16.0;
			if (distance >= 0.05 && distance <= 0.07)             // SDS = 0.06 cm
				f->data[3] += p->weight / 24.0;
			if (distance >= 0.07 && distance <= 0.09)             // SDS = 0.08 cm
				f->data[4] += p->weight / 32.0;
		}
		else
		{
			distance = sqrt(p->x * p->x + p->y * p->y);
			//all circular
			/*
			for (int i = 1; i <= NUM_OF_DETECTOR; i++)
			{
			if ((distance > (f->position[i] - f->radius[i])) && (distance <= (f->position[i] + f->radius[i])))
			{
			//record circular instead of a fiber area
			f->data[i] += p->weight;
			}
			}
			*/
			//fiber only
			for (int i = 1; i <= NUM_OF_DETECTOR; i++)
			{
				if ((distance >= (f->position[i] - f->radius[i])) && (distance <= (f->position[i] + f->radius[i])))
				{
					float temp;
					temp = (distance*distance + f->position[i] * f->position[i] - f->radius[i] * f->radius[i]) / (2 * distance*f->position[i]);
					// check for rounding error!
					if (temp > 1.0f)
						temp = 1.0f;

					f->data[i] += p->weight  * acos(temp) * RPI;
					f->photon_detected[i] = true;

				}
			}
		}
	}
	return;
}

int InitDCMem(SimulationStruct* sim)
{
	// Copy num_photons_dc to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(n_layers_dc), &(sim->n_layers), sizeof(unsigned int));

	// Copy start_weight_dc to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(start_weight_dc), &(sim->start_weight), sizeof(unsigned int));

	// Copy layer data to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(layers_dc), sim->layers, (sim->n_layers + 2) * sizeof(LayerStruct));

	// Copy num_photons_dc to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(num_photons_dc), &(sim->number_of_photons), sizeof(unsigned long long));

	return 0;
}

int InitMemStructs(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim, char* fiber1_position) //Wang modified
{
	// Allocate p on the device!!
	hipMalloc((void**)&DeviceMem->p, NUM_THREADS * sizeof(PhotonStruct));

	// Allocate thread_active on the device and host
	HostMem->thread_active = (unsigned int*)malloc(NUM_THREADS * sizeof(unsigned int));
	if (HostMem->thread_active == NULL) { printf("Error allocating HostMem->thread_active"); exit(1); }
	for (int i = 0; i<NUM_THREADS; i++)HostMem->thread_active[i] = 1u;

	hipMalloc((void**)&DeviceMem->thread_active, NUM_THREADS * sizeof(unsigned int));
	hipMemcpy(DeviceMem->thread_active, HostMem->thread_active, NUM_THREADS * sizeof(unsigned int), hipMemcpyHostToDevice);

	//Allocate num_launched_photons on the device and host
	HostMem->num_terminated_photons = (unsigned long long*) malloc(sizeof(unsigned long long));
	if (HostMem->num_terminated_photons == NULL) { printf("Error allocating HostMem->num_terminated_photons"); exit(1); }
	*HostMem->num_terminated_photons = 0;

	hipMalloc((void**)&DeviceMem->num_terminated_photons, sizeof(unsigned long long));
	hipMemcpy(DeviceMem->num_terminated_photons, HostMem->num_terminated_photons, sizeof(unsigned long long), hipMemcpyHostToDevice);

	//Allocate and initialize fiber f on the device and host
	HostMem->f = (Fibers*)malloc(NUM_THREADS * sizeof(Fibers));
	hipMalloc((void**)&DeviceMem->f, NUM_THREADS * sizeof(Fibers));
	fiber_initialization(HostMem->f, atof(fiber1_position)); //Wang modified
	hipMemcpy(DeviceMem->f, HostMem->f, NUM_THREADS * sizeof(Fibers), hipMemcpyHostToDevice);

	//Allocate states on the device and host
	hipMalloc((void**)&DeviceMem->state, NUM_THREADS * sizeof(hiprandState));

	return 1;
}

void FreeMemStructs(MemStruct* HostMem, MemStruct* DeviceMem)
{
	free(HostMem->thread_active);
	free(HostMem->num_terminated_photons);
	free(HostMem->f);

	hipFree(DeviceMem->thread_active);
	hipFree(DeviceMem->num_terminated_photons);
	hipFree(DeviceMem->f);
	hipFree(DeviceMem->state);
}